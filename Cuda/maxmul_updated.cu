#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "maxmul.h"
 
//This file is being created for experiment purpose, in case we decide to do
//some testing our the main logic. To be removed later.
__global__ void matrixMultiplication(float *A, float *B, float *C, int size) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < size && col < size) {
        float sum = 0.0;
        for (int k = 0; k < size; k++) {
            // printf("row = %d, col = %d", row * size + k, k * size + col);
            sum += A[row * size + k] * B[k * size + col];
        }
        C[row * size + col] = sum;
    }
}

extern "C" {

   void maxmulVector(float *A, float *B, float *C, int size) {
    float *d_A, *d_B, *d_C;
    clock_t startTime = clock();
    // printf("Point1");
    // Allocate memory on the GPU for A, B, and C
    hipMalloc((void**)&d_A, size * size * sizeof(float));
    hipMalloc((void**)&d_B, size * size * sizeof(float));
    hipMalloc((void**)&d_C, size * size * sizeof(float));
    // printf("Point2");
    // Copy A and B from host to device
    hipMemcpy(d_A, A, size * size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size * size * sizeof(float), hipMemcpyHostToDevice);
    // printf("Point3");
    // Set grid and block dimensions
    dim3 dimGrid((size + 15) / 16, (size + 15) / 16);
    dim3 dimBlock(16, 16);

    // printf("Point1=4");
    // Launch the CUDA kernel to perform matrix multiplication

    
    matrixMultiplication<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, size);
    
    // printf("Point5");
    // Copy the result C from device to host
    hipMemcpy(C, d_C, size * size * sizeof(float), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipDeviceSynchronize();
        clock_t endTime = clock();
        double elapsedTime = (double)(endTime - startTime) / CLOCKS_PER_SEC;
        printf("\nGPU Execution Time: %f seconds\n", elapsedTime);
}

}